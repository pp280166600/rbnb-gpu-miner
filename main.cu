#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdint.h>
#include <stdio.h>
#include <io.h>
#include <process.h>
#include "common.cuh"
#include "getopt.cuh"
#include "sha3.cuh"

#define BLOCKS 32
#define THREADS 256

#define N 1000000
struct Result
{
    char id[65];
};

__device__ uint8_t dev_wanted_signature[4] = {0x0, 0x0, 0x0, 0x0};
__device__ uint8_t dev_wanted_signature2[3] = {0x99, 0x99, 0x99};

__global__ void init_signature(uint32_t *fn_sig)
{
    dev_wanted_signature[0] = *fn_sig >> 24;
    dev_wanted_signature[1] = ((*fn_sig >> 16) & 0xff);
    dev_wanted_signature[2] = ((*fn_sig >> 8) & 0xff);
    dev_wanted_signature[3] = ((*fn_sig >> 0) & 0xff);
}

__host__ __device__ unsigned char hex_char_to_char(char c)
{
    if (c >= '0' && c <= '9')
    {
        return (unsigned char)(c - '0');
    }
    else if (c >= 'a' && c <= 'f')
    {
        return (unsigned char)(c - 'a' + 10);
    }
    else if (c >= 'A' && c <= 'F')
    {
        return (unsigned char)(c - 'A' + 10);
    }
    else
    {
        return 0;
    }
}

// 将十六进制字符串转换为字节数组
__host__ __device__ void hex_string_to_char_array(char hex_string[], unsigned char char_array[])
{
    size_t len = _strlen(hex_string);
    size_t byte_len = len / 2;

    for (size_t i = 0; i < byte_len; ++i)
    {
        char_array[i] = (hex_char_to_char(hex_string[i * 2]) << 4) | hex_char_to_char(hex_string[i * 2 + 1]);
    }
}

// 避免内存重叠版的memcpy
__device__ void *_memcpy(void *dst, const void *src, unsigned int count)
{
    void *ret = dst;
    if (dst <= src || (char *)dst >= ((char *)src + count)) //
    {
        while (count--)
        {
            *(char *)dst = *(char *)src;
            dst = (char *)dst + 1;
            src = (char *)src + 1;
        }
    }
    else
    {
        dst = (char *)dst + count - 1;
        src = (char *)src + count - 1;
        while (count--)
        {
            *(char *)dst = *(char *)src;
            dst = (char *)dst - 1;
            src = (char *)src - 1;
        }
    }
    return ret;
}

__global__ void calculate(char *address, char *challenge_value, Result *results)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock64() + tid, tid, 0, &state);
    char id[65];
    char res[193];
    char challenge_hex[] = "72424e4200000000000000000000000000000000000000000000000000000000000000000000000000000000";
    const size_t len = 96;
    unsigned char data[len];
    char hex_array[] = {'0', '1', '2', '3', '4', '5', '6', '7',
                        '8', '9', 'a', 'b', 'c', 'd', 'e', 'f'};
    for (int i = 0; i < N; i++)
    {
        uint8_t hash[64];
        memset(id, 0, 64);
        for (int k = 0; k < 64; k++)
        {
            int block = (hiprand(&state) % 16);
            id[k] = hex_array[block];
        }
        memset(data, 0, len);
        memset(res, 0, 193);
        id[64] = '\0';
        _memcpy(&res, id, _strlen(id));
        _memcpy(&res[_strlen(res)], challenge_hex, _strlen(challenge_hex));
        _memcpy(&res[_strlen(res)], address, _strlen(address));
        hex_string_to_char_array(res, data);
        sha3_return_t ok = sha3_HashBuffer(256, SHA3_FLAGS_KECCAK, data,
                                           len, hash, 64);
        if (ok != 0)
        {
            printf("bad params\n");
            return;
        }
        if (hash[0] == dev_wanted_signature2[0] &&
            hash[1] == dev_wanted_signature2[1] &&
            hash[2] == dev_wanted_signature2[2])
        {
            _memcpy(&results[tid].id, id, _strlen(id));
            printf("Tid: %d  Hex: %s\n", tid, id);
            return;
        }
    }
}

int main(int argc, char **argv)
{
    int opt;
    char *avalue = NULL; // 
    char *mvalue = NULL; // 
    while ((opt = getopt(argc, argv, "a:m:")) != -1)
    {
        switch (opt)
        {
        case 'm':
            mvalue = optarg;
            break;
        case 'a':
            avalue = optarg;
            break;
        default: /* '?' */
            usage(argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    if (mvalue == NULL || avalue == NULL)
    {
        usage(argv[0]);
        exit(EXIT_FAILURE);
    }
    const char *filename = "result.txt";
    FILE *file = fopen(filename, "r+");
    if (file == NULL)
    {
        file = fopen(filename, "w+");
    }
    fseek(file, 0, SEEK_END);

    char *dev_m, *dev_a;
    Result *host_results = (Result *)malloc(BLOCKS * THREADS * sizeof(Result));
    Result *device_data;
    HANDLE_ERROR(hipMalloc((void **)&device_data, BLOCKS * THREADS * sizeof(Result)));
    HANDLE_ERROR(hipMalloc((void **)&dev_m, _strlen(mvalue) * sizeof(char)));
    HANDLE_ERROR(hipMalloc((void **)&dev_a, _strlen(avalue) * sizeof(char)));
    HANDLE_ERROR(hipMemcpy(dev_m, mvalue, _strlen(mvalue) * sizeof(char),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_a, avalue, _strlen(avalue) * sizeof(char),
                            hipMemcpyHostToDevice));
    calculate<<<BLOCKS, THREADS>>>(dev_a, dev_m, (Result *)device_data);
    hipDeviceSynchronize(); // not important
    HANDLE_ERROR(hipMemcpy(host_results, device_data, (BLOCKS * THREADS * sizeof(Result)), hipMemcpyDeviceToHost));
    for (int i = 0; i < BLOCKS * THREADS; i++)
    {

        if (_strlen(host_results[i].id) > 0)
        {
            fprintf(file, "0x%s,0x%s\n", host_results[i].id, avalue);
        }
    }
    HANDLE_ERROR(hipFree(dev_m));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(device_data));
    free(host_results);
    printf("success all, result for result.txt\n");
    exit(EXIT_SUCCESS);
}
